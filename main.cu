#include <iostream>
#include "interp2.cuh"


int main() {


    int nX = 2;
    int nY = 2;
    int points = 5;

    int sizeX = sizeof(xt) * nX;
    int sizeY = sizeof(yt) * nY;
    int sizeZ = sizeof(zt) * nX * nY;
    int sizeXq= sizeof(xt) * points;
    int sizeYq= sizeof(yt) * points;
    int sizeZq= sizeof(zt) * points;

    xt *X = (xt *) malloc(sizeX);
    yt *Y = (yt *) malloc(sizeY);;
    zt *Z = (zt *) malloc(sizeZ);;
    xt *Xq = (xt *) malloc(sizeXq);
    yt *Yq = (yt *) malloc(sizeYq);
    zt *Zq = (zt *) malloc(sizeZq);

    float *dX;
    float *dY;
    float *dZ;
    float *dXq;
    float *dYq;
    float *dZq;

    X[0] = 0;
    X[1] = 1;

    Y[0] = 0;
    Y[1] = 1;

    Z[0] = 1;
    Z[1] = 1;
    Z[2] = 2;
    Z[3] = 2;


    hipMalloc(&dX, sizeX);
    hipMalloc(&dY, sizeY);
    hipMalloc(&dZ, sizeZ);
    hipMalloc(&dXq, sizeXq);
    hipMalloc(&dYq, sizeYq);
    hipMalloc(&dZq, sizeZq);

    hipMemcpy(dX, X, sizeX, hipMemcpyHostToDevice);
    hipMemcpy(dY, Y, sizeY, hipMemcpyHostToDevice);
    hipMemcpy(dZ, Z, sizeZ, hipMemcpyHostToDevice);
    hipMemcpy(dXq, Xq, sizeXq, hipMemcpyHostToDevice);
    hipMemcpy(dYq, Yq, sizeYq, hipMemcpyHostToDevice);
    hipMemcpy(dZq, Zq, sizeZq, hipMemcpyHostToDevice);

    for( int i = 0; i < 5; i++)
    {
        Xq[i] = (xt)i/4;
        Yq[i] = (yt)i/4;
    }



    int threadsPerBlock = 256;
    int blocksPerGrid =
            (points + threadsPerBlock - 1) / threadsPerBlock;
    device::interp2<<<blocksPerGrid, threadsPerBlock>>>(dX, nX,
                                                        dY, nY,
                                                        dZ,
                                                        dXq, dYq,
                                                        dZq, points);


    hipMemcpy(dZq, Zq, sizeZq, hipMemcpyDeviceToHost);

    for (int i =0; i<5; i++)
    {
        std::cout<<Xq[i]<<" "<<Yq[i]<<" "<<Zq[i]<<std::endl;
    }


    return 0;
}
